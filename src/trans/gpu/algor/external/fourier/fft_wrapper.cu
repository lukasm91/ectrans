#include "hip/hip_runtime.h"
// (C) Copyright 2022- NVIDIA.
//
// This software is licensed under the terms of the Apache Licence Version 2.0
// which can be obtained at http://www.apache.org/licenses/LICENSE-2.0.
// In applying this licence, ECMWF does not waive the privileges and immunities
// granted to it by virtue of its status as an intergovernmental organisation
// nor does it submit to any jurisdiction.

#include "hipfft/hipfft.h"
#include "stdio.h"
#include <iostream>
#include <unordered_map>
#include <vector>

#include "../growing_allocator.h"

static const char *_cudaGetErrorEnum(hipfftResult error) {
  switch (error) {
  case HIPFFT_SUCCESS:
    return "HIPFFT_SUCCESS";

  case HIPFFT_INVALID_PLAN:
    return "HIPFFT_INVALID_PLAN";

  case HIPFFT_ALLOC_FAILED:
    return "HIPFFT_ALLOC_FAILED";

  case HIPFFT_INVALID_TYPE:
    return "HIPFFT_INVALID_TYPE";

  case HIPFFT_INVALID_VALUE:
    return "HIPFFT_INVALID_VALUE";

  case HIPFFT_INTERNAL_ERROR:
    return "HIPFFT_INTERNAL_ERROR";

  case HIPFFT_EXEC_FAILED:
    return "HIPFFT_EXEC_FAILED";

  case HIPFFT_SETUP_FAILED:
    return "HIPFFT_SETUP_FAILED";

  case HIPFFT_INVALID_SIZE:
    return "HIPFFT_INVALID_SIZE";

  case HIPFFT_UNALIGNED_DATA:
    return "HIPFFT_UNALIGNED_DATA";
  }

  return "<unknown>";
}
#define CUDA_CHECK(e)                                                          \
  {                                                                            \
    hipError_t err = (e);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error: %s, line %d, %s: %s\n", __FILE__, __LINE__, \
              #e, hipGetErrorString(err));                                    \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

#define CUFFT_CHECK(e)                                                         \
  {                                                                            \
    hipfftResult_t err = (e);                                                   \
    if (err != HIPFFT_SUCCESS) {                                                \
      fprintf(stderr, "CUFFT error: %s, line %d, %s: %s\n", __FILE__,          \
              __LINE__, #e, _cudaGetErrorEnum(err));                           \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

namespace {
struct Double {
  using real = double;
  using cmplx = hipfftDoubleComplex;
};
struct Float {
  using real = float;
  using cmplx = hipfftComplex;
};

// kfield -> handles
template <class Type, hipfftType Direction> auto &get_fft_plan_cache() {
  static std::unordered_map<int, std::vector<hipfftHandle>> fftPlansCache;
  return fftPlansCache;
}
// kfield -> graphs
template <class Type, hipfftType Direction> auto &get_graph_cache() {
  static std::unordered_map<int, hipGraphExec_t> graphCache;
  return graphCache;
}
// kfield -> ptrs
template <class Type, hipfftType Direction> auto &get_ptr_cache() {
  using real = typename Type::real;
  using cmplx = typename Type::cmplx;
  static std::unordered_map<int, std::pair<real *, cmplx *>> ptrCache;
  return ptrCache;
}

template <class Type, hipfftType Direction>
void free_fft_cache(float *, size_t) {
  get_graph_cache<Type, Direction>().clear();
  get_ptr_cache<Type, Direction>().clear();
}

static constexpr bool run_fft_parallel = false;

template <class Type, hipfftType Direction>
size_t plan_fft(int kfield, int *loens, int *offsets, int nfft, bool allocate) {

  constexpr bool is_forward = Direction == HIPFFT_R2C || Direction == HIPFFT_D2Z;

  auto &fftPlansCache =
      get_fft_plan_cache<Type, Direction>(); // kfield -> handles
  auto fftPlans = fftPlansCache.find(kfield);
  if (fftPlans == fftPlansCache.end()) {
    // the fft plans do not exist yet
    std::vector<hipfftHandle> newPlans;
    newPlans.resize(nfft);
    for (int i = 0; i < nfft; ++i) {
      int nloen = loens[i];

      hipfftHandle plan;
      CUFFT_CHECK(hipfftCreate(&plan));
      if (!allocate) {
        CUFFT_CHECK(hipfftSetAutoAllocation(plan, false));
      }
      int dist = offsets[i + 1] - offsets[i];
      int embed[] = {1};
      size_t worksize;
      CUFFT_CHECK(hipfftMakePlanMany(
          plan, 1, &nloen, embed, 1, is_forward ? dist : dist / 2, embed, 1,
          is_forward ? dist / 2 : dist, Direction, kfield, &worksize));
      newPlans[i] = plan;
    }
    fftPlans = fftPlansCache.insert({kfield, newPlans}).first;
  }

  size_t total_worksize = 0;
  for (auto const &plan : fftPlans->second) {
    size_t local_worksize;
    CUFFT_CHECK(hipfftGetSize(plan, &local_worksize));
    if (run_fft_parallel)
      total_worksize += local_worksize;
    else
      total_worksize = max(local_worksize, total_worksize);
  }
  return total_worksize;
}
template <class Type, hipfftType Direction>
void execute_fft(typename Type::real *data_real,
                 typename Type::cmplx *data_complex, int kfield, int *loens,
                 int *offsets, int nfft, void *growing_allocator,
                 void *buffer) {

  growing_allocator_register_free_c(growing_allocator,
                                    free_fft_cache<Type, Direction>);

  using real = typename Type::real;
  using cmplx = typename Type::cmplx;

  // if the pointers are changed, we need to update the graph
  auto &ptrCache = get_ptr_cache<Type, Direction>();     // kfield -> ptrs
  auto &graphCache = get_graph_cache<Type, Direction>(); // kfield -> graphs

  auto ptrs = ptrCache.find(kfield);
  if (ptrs != ptrCache.end() && (ptrs->second.first != data_real ||
                                 ptrs->second.second != data_complex)) {
    // the plan is cached, but the pointers are not correct. we remove and
    // delete the graph, but we keep the FFT plans, if this happens more often,
    // we should cache this...
    std::cout << "WARNING FFT: POINTER CHANGE --> THIS MIGHT BE SLOW"
              << std::endl;
    CUDA_CHECK(hipGraphExecDestroy(graphCache[kfield]));
    graphCache.erase(kfield);
    ptrCache.erase(kfield);
  }

  auto graph = graphCache.find(kfield);
  if (graph == graphCache.end()) {
    // this graph does not exist yet

    auto &fftPlansCache =
        get_fft_plan_cache<Type, Direction>(); // kfield -> handles
    auto fftPlans = fftPlansCache.find(kfield);
    if (fftPlans == fftPlansCache.end())
      exit(EXIT_FAILURE);

    if (buffer) {
      size_t total_worksize = 0;
      for (auto const &plan : fftPlans->second) {
        size_t local_worksize;
        CUFFT_CHECK(hipfftGetSize(plan, &local_worksize));
        CUFFT_CHECK(hipfftSetWorkArea(plan, (char *)buffer + total_worksize));
        if (run_fft_parallel)
          total_worksize += local_worksize;
      }
    }

    // create a temporary stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    for (auto &plan : fftPlans->second) // set the streams
      CUFFT_CHECK(hipfftSetStream(plan, stream));

    // now create the cuda graph
    hipGraph_t new_graph;
    if (run_fft_parallel)
      hipGraphCreate(&new_graph, 0);
    else
      CUDA_CHECK(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
    for (int i = 0; i < nfft; ++i) {
      size_t offset = (size_t)offsets[i];
      real *data_real_l = &data_real[kfield * offset];
      cmplx *data_complex_l = &data_complex[kfield * offset / 2];
      if (run_fft_parallel)
        CUDA_CHECK(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
      if constexpr (Direction == HIPFFT_R2C)
        CUFFT_CHECK(
            hipfftExecR2C(fftPlans->second[i], data_real_l, data_complex_l))
      else if constexpr (Direction == HIPFFT_C2R)
        CUFFT_CHECK(
            hipfftExecC2R(fftPlans->second[i], data_complex_l, data_real_l))
      else if constexpr (Direction == HIPFFT_D2Z)
        CUFFT_CHECK(
            hipfftExecD2Z(fftPlans->second[i], data_real_l, data_complex_l))
      else if constexpr (Direction == HIPFFT_Z2D)
        CUFFT_CHECK(
            hipfftExecZ2D(fftPlans->second[i], data_complex_l, data_real_l));
      if (run_fft_parallel) {
        hipGraph_t my_graph;
        CUDA_CHECK(hipStreamEndCapture(stream, &my_graph));
        hipGraphNode_t my_node;
        CUDA_CHECK(hipGraphAddChildGraphNode(&my_node, new_graph, nullptr, 0,
                                              my_graph));
      }
    }
    if (!run_fft_parallel)
      CUDA_CHECK(hipStreamEndCapture(stream, &new_graph));
    hipGraphExec_t instance;
    CUDA_CHECK(hipGraphInstantiate(&instance, new_graph, NULL, NULL, 0));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipGraphDestroy(new_graph));

    graphCache.insert({kfield, instance});
    ptrCache.insert({kfield, std::make_pair(data_real, data_complex)});
  }

  CUDA_CHECK(hipGraphLaunch(graphCache.at(kfield), 0));
  CUDA_CHECK(hipDeviceSynchronize());
}
} // namespace

extern "C" {
void execute_dir_fft_float(float *data_real, hipfftComplex *data_complex,
                           int kfield, int *loens, int *offsets, int nfft,
                           void *growing_allocator, void *buffer) {
  execute_fft<Float, HIPFFT_R2C>(data_real, data_complex, kfield, loens, offsets,
                                nfft, growing_allocator, buffer);
}
void execute_inv_fft_float(hipfftComplex *data_complex, float *data_real,
                           int kfield, int *loens, int *offsets, int nfft,
                           void *growing_allocator, void *buffer) {
  execute_fft<Float, HIPFFT_C2R>(data_real, data_complex, kfield, loens, offsets,
                                nfft, growing_allocator, buffer);
}
void execute_dir_fft_double(double *data_real, hipfftDoubleComplex *data_complex,
                            int kfield, int *loens, int *offsets, int nfft,
                            void *growing_allocator, void *buffer) {
  execute_fft<Double, HIPFFT_D2Z>(data_real, data_complex, kfield, loens,
                                 offsets, nfft, growing_allocator, buffer);
}
void execute_inv_fft_double(hipfftDoubleComplex *data_complex, double *data_real,
                            int kfield, int *loens, int *offsets, int nfft,
                            void *growing_allocator, void *buffer) {
  execute_fft<Double, HIPFFT_Z2D>(data_real, data_complex, kfield, loens,
                                 offsets, nfft, growing_allocator, buffer);
}
size_t plan_dir_fft_float(int kfield, int *loens, int *offsets, int nfft,
                          int allocate) {
  return plan_fft<Float, HIPFFT_R2C>(kfield, loens, offsets, nfft, allocate);
}
size_t plan_inv_fft_float(int kfield, int *loens, int *offsets, int nfft,
                          int allocate) {
  return plan_fft<Float, HIPFFT_C2R>(kfield, loens, offsets, nfft, allocate);
}
size_t plan_dir_fft_double(int kfield, int *loens, int *offsets, int nfft,
                           int allocate) {
  return plan_fft<Double, HIPFFT_D2Z>(kfield, loens, offsets, nfft, allocate);
}
size_t plan_inv_fft_double(int kfield, int *loens, int *offsets, int nfft,
                           int allocate) {
  return plan_fft<Double, HIPFFT_Z2D>(kfield, loens, offsets, nfft, allocate);
}
}
