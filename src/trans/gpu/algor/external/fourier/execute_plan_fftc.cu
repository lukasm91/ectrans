
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "stdio.h"
#include <iostream>
#include <unordered_map>
#include <vector>

static const char *_cudaGetErrorEnum(hipfftResult error) {
  switch (error) {
  case HIPFFT_SUCCESS:
    return "CUFFT_SUCCESS";

  case HIPFFT_INVALID_PLAN:
    return "CUFFT_INVALID_PLAN";

  case HIPFFT_ALLOC_FAILED:
    return "CUFFT_ALLOC_FAILED";

  case HIPFFT_INVALID_TYPE:
    return "CUFFT_INVALID_TYPE";

  case HIPFFT_INVALID_VALUE:
    return "CUFFT_INVALID_VALUE";

  case HIPFFT_INTERNAL_ERROR:
    return "CUFFT_INTERNAL_ERROR";

  case HIPFFT_EXEC_FAILED:
    return "CUFFT_EXEC_FAILED";

  case HIPFFT_SETUP_FAILED:
    return "CUFFT_SETUP_FAILED";

  case HIPFFT_INVALID_SIZE:
    return "CUFFT_INVALID_SIZE";

  case HIPFFT_UNALIGNED_DATA:
    return "CUFFT_UNALIGNED_DATA";
  }

  return "<unknown>";
}
#define CUDA_CHECK(e)                                                          \
  {                                                                            \
    hipError_t err = (e);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error: %s, line %d, %s: %s\n", __FILE__, __LINE__, \
              #e, hipGetErrorString(err));                                    \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

#define CUFFT_CHECK(e) { \
	hipfftResult_t err = (e); \
	if (err != HIPFFT_SUCCESS) \
	{ \
		fprintf(stderr, "CUFFT error: %s, line %d, %s: %s\n", \
			__FILE__, __LINE__, #e, _cudaGetErrorEnum(err)); \
		exit(EXIT_FAILURE); \
	} \
}

extern void *planWorkspace;

extern "C" void
#ifdef TRANS_SINGLE
execute_plan_fftc_(cufftHandle *PLANp, int *ISIGNp, cufftComplex *data_in,
                   cufftComplex *data_out)
#else
execute_plan_fftc_(hipfftHandle *PLANp, int *ISIGNp, hipfftDoubleComplex *data_in,
                   hipfftDoubleComplex *data_out)
#endif
{
  hipfftHandle plan = *PLANp;
  int ISIGN = *ISIGNp;

  CUFFT_CHECK(hipfftSetWorkArea(plan, planWorkspace));

  if (ISIGN == -1) {
#ifdef TRANS_SINGLE
    CUFFT_CHECK(cufftExecR2C(plan, (cufftReal *)data_in, data_out));
#else
    CUFFT_CHECK(hipfftExecD2Z(plan, (hipfftDoubleReal *)data_in, data_out));
#endif
  } else if (ISIGN == 1) {
#ifdef TRANS_SINGLE
    CUFFT_CHECK(cufftExecC2R(plan, data_in, (cufftReal *)data_out));
#else
    CUFFT_CHECK(hipfftExecZ2D(plan, data_in, (hipfftDoubleReal *)data_out));
#endif
  } else {
    abort();
  }
}

namespace {
struct Double {
  using real = double;
  using cmplx = hipfftDoubleComplex;
};
struct Float {
  using real = float;
  using cmplx = hipfftComplex;
};
}
template <class Type, hipfftType Direction>
void execute_fft(typename Type::real *data_real, typename Type::cmplx *data_complex,
        int kfield, int *loens, int *offsets, int nfft) {
  using real = typename Type::real;
  using cmplx = typename Type::cmplx;

  /* static std::unordered_map<int, void *> allocationCache; // nloens -> ptr */
  static std::unordered_map<int, std::vector<hipfftHandle>> fftPlansCache; // kfield -> handles
  static std::unordered_map<int, hipGraphExec_t> graphCache; // kfield -> graphs

  // if the pointers are changed, we need to update the graph
  static std::unordered_map<int, std::pair<real *, cmplx *>> ptrCache; // kfield -> ptrs

  auto ptrs = ptrCache.find(kfield);
  if (ptrs != ptrCache.end() && (
              ptrs->second.first != data_real || ptrs->second.second != data_complex)) {
      // the plan is cached, but the pointers are not correct. we remove and delete the graph,
      // but we keep the FFT plans, if this happens more often, we should cache this...
      std::cout << "WARNING: POINTER CHANGE --> THIS MIGHT BE SLOW" << std::endl;
      CUDA_CHECK(hipGraphExecDestroy(graphCache[kfield]));
      graphCache.erase(kfield);
      ptrCache.erase(kfield);
  }

  auto graph = graphCache.find(kfield);
  if (graph == graphCache.end()) {
      // this graph does not exist yet

      auto fftPlans = fftPlansCache.find(kfield);
      if (fftPlans == fftPlansCache.end()) {
          // the fft plans do not exist yet
          std::vector<hipfftHandle> newPlans;
          newPlans.resize(nfft);
          for (int i = 0; i < nfft; ++i) {
            int nloen = loens[i];

            hipfftHandle plan;
            CUFFT_CHECK(hipfftCreate(&plan));
            int dist = 1;
            int embed[] = {1};
            CUFFT_CHECK(hipfftPlanMany(&plan, 1, &nloen, embed, kfield, dist, embed,
                                      kfield, dist, Direction, kfield));
            newPlans[i] = plan;
          }
          fftPlansCache.insert({kfield, newPlans});
      }
      fftPlans = fftPlansCache.find(kfield);

      // create a temporary stream
      hipStream_t stream;
      CUDA_CHECK(hipStreamCreate(&stream));

      for (auto &plan : fftPlans->second) // set the streams
        CUFFT_CHECK(hipfftSetStream(plan, stream));

      // now create the cuda graph
      hipGraph_t new_graph;
      hipGraphCreate(&new_graph, 0);
      for (int i = 0; i < nfft; ++i) {
        int offset = offsets[i];
        real *data_real_l = &data_real[kfield * offset];
        cmplx *data_complex_l = &data_complex[kfield * offset / 2];
        CUDA_CHECK(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
        if constexpr(Direction == HIPFFT_R2C)
          CUFFT_CHECK(hipfftExecR2C(fftPlans->second[i], data_real_l, data_complex_l))
        else if constexpr(Direction == HIPFFT_C2R)
          CUFFT_CHECK(hipfftExecC2R(fftPlans->second[i], data_complex_l, data_real_l))
        else if constexpr(Direction == HIPFFT_D2Z)
          CUFFT_CHECK(hipfftExecD2Z(fftPlans->second[i], data_real_l, data_complex_l))
        else if constexpr(Direction == HIPFFT_Z2D)
          CUFFT_CHECK(hipfftExecZ2D(fftPlans->second[i], data_complex_l, data_real_l));
        hipGraph_t my_graph;
        CUDA_CHECK(hipStreamEndCapture(stream, &my_graph));
        hipGraphNode_t my_node;
        CUDA_CHECK(hipGraphAddChildGraphNode(&my_node, new_graph, nullptr, 0, my_graph));
      }
      hipGraphExec_t instance;
      CUDA_CHECK(hipGraphInstantiate(&instance, new_graph, NULL, NULL, 0));
      CUDA_CHECK(hipStreamDestroy(stream));
      CUDA_CHECK(hipGraphDestroy(new_graph));

      graphCache.insert({kfield, instance});
      ptrCache.insert({kfield, std::make_pair(data_real, data_complex)});
  }

  CUDA_CHECK(hipGraphLaunch(graphCache.at(kfield), 0));
      /* for (int i = 0; i < nfft; ++i) { */
        /* int nloen = loens[i]; */

        /* cufftHandle plan; */
        /* CUFFT_CHECK(cufftCreate(&plan)); */
        /* int dist = 1; */
        /* int embed[] = {1}; */
        /* CUFFT_CHECK(cufftPlanMany(&plan, 1, &nloen, embed, kfield, dist, embed, */
                                  /* kfield, dist, Direction, kfield)); */
        /* int offset = offsets[i]; */
        /* real *data_real_l = &data_real[kfield * offset]; */
        /* cmplx *data_complex_l = &data_complex[kfield * offset / 2]; */
        /* if (Direction == CUFFT_R2C) */
          /* CUFFT_CHECK(cufftExecR2C(plan, data_real_l, data_complex_l)) */
        /* else */
          /* CUFFT_CHECK(cufftExecC2R(plan, data_complex_l, data_real_l)); */
        /* CUFFT_CHECK(cufftDestroy(plan)); */
      /* } */
  CUDA_CHECK(hipDeviceSynchronize());
}
template <class Type, hipfftType Direction>
void execute_fft_new(typename Type::real *data_real, typename Type::cmplx *data_complex,
        int kfield, int *loens, int *offsets, int nfft) {
  using real = typename Type::real;
  using cmplx = typename Type::cmplx;

  /* static std::unordered_map<int, void *> allocationCache; // nloens -> ptr */
  static std::unordered_map<int, std::vector<hipfftHandle>> fftPlansCache; // kfield -> handles
  static std::unordered_map<int, hipGraphExec_t> graphCache; // kfield -> graphs

  // if the pointers are changed, we need to update the graph
  static std::unordered_map<int, std::pair<real *, cmplx *>> ptrCache; // kfield -> ptrs

  auto ptrs = ptrCache.find(kfield);
  if (ptrs != ptrCache.end() && (
              ptrs->second.first != data_real || ptrs->second.second != data_complex)) {
      // the plan is cached, but the pointers are not correct. we remove and delete the graph,
      // but we keep the FFT plans, if this happens more often, we should cache this...
      std::cout << "WARNING: POINTER CHANGE --> THIS MIGHT BE SLOW" << std::endl;
      CUDA_CHECK(hipGraphExecDestroy(graphCache[kfield]));
      graphCache.erase(kfield);
      ptrCache.erase(kfield);
  }

  auto graph = graphCache.find(kfield);
  if (graph == graphCache.end()) {
      // this graph does not exist yet

      auto fftPlans = fftPlansCache.find(kfield);
      if (fftPlans == fftPlansCache.end()) {
          // the fft plans do not exist yet
          std::vector<hipfftHandle> newPlans;
          newPlans.resize(nfft);
          for (int i = 0; i < nfft; ++i) {
            int nloen = loens[i];

            hipfftHandle plan;
            CUFFT_CHECK(hipfftCreate(&plan));
            int dist = offsets[i+1] - offsets[i];
            int embed[] = {1};
            CUFFT_CHECK(hipfftPlanMany(&plan, 1, &nloen, embed, 1, dist, embed,
                                      1, dist / 2, Direction, kfield));
            newPlans[i] = plan;
          }
          fftPlansCache.insert({kfield, newPlans});
      }
      fftPlans = fftPlansCache.find(kfield);

      // create a temporary stream
      hipStream_t stream;
      CUDA_CHECK(hipStreamCreate(&stream));

      for (auto &plan : fftPlans->second) // set the streams
        CUFFT_CHECK(hipfftSetStream(plan, stream));

      // now create the cuda graph
      hipGraph_t new_graph;
      hipGraphCreate(&new_graph, 0);
      for (int i = 0; i < nfft; ++i) {
        int offset = offsets[i];
        real *data_real_l = &data_real[kfield * offset];
        cmplx *data_complex_l = &data_complex[kfield * offset / 2];
        CUDA_CHECK(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
        if constexpr(Direction == HIPFFT_R2C)
          CUFFT_CHECK(hipfftExecR2C(fftPlans->second[i], data_real_l, data_complex_l))
        else if constexpr(Direction == HIPFFT_C2R)
          CUFFT_CHECK(hipfftExecC2R(fftPlans->second[i], data_complex_l, data_real_l))
        else if constexpr(Direction == HIPFFT_D2Z)
          CUFFT_CHECK(hipfftExecD2Z(fftPlans->second[i], data_real_l, data_complex_l))
        else if constexpr(Direction == HIPFFT_Z2D)
          CUFFT_CHECK(hipfftExecZ2D(fftPlans->second[i], data_complex_l, data_real_l));
        hipGraph_t my_graph;
        CUDA_CHECK(hipStreamEndCapture(stream, &my_graph));
        hipGraphNode_t my_node;
        CUDA_CHECK(hipGraphAddChildGraphNode(&my_node, new_graph, nullptr, 0, my_graph));
      }
      hipGraphExec_t instance;
      CUDA_CHECK(hipGraphInstantiate(&instance, new_graph, NULL, NULL, 0));
      CUDA_CHECK(hipStreamDestroy(stream));
      CUDA_CHECK(hipGraphDestroy(new_graph));

      graphCache.insert({kfield, instance});
      ptrCache.insert({kfield, std::make_pair(data_real, data_complex)});
  }

  CUDA_CHECK(hipGraphLaunch(graphCache.at(kfield), 0));
      /* for (int i = 0; i < nfft; ++i) { */
        /* int nloen = loens[i]; */

        /* cufftHandle plan; */
        /* CUFFT_CHECK(cufftCreate(&plan)); */
        /* int dist = 1; */
        /* int embed[] = {1}; */
        /* CUFFT_CHECK(cufftPlanMany(&plan, 1, &nloen, embed, kfield, dist, embed, */
                                  /* kfield, dist, Direction, kfield)); */
        /* int offset = offsets[i]; */
        /* real *data_real_l = &data_real[kfield * offset]; */
        /* cmplx *data_complex_l = &data_complex[kfield * offset / 2]; */
        /* if (Direction == CUFFT_R2C) */
          /* CUFFT_CHECK(cufftExecR2C(plan, data_real_l, data_complex_l)) */
        /* else */
          /* CUFFT_CHECK(cufftExecC2R(plan, data_complex_l, data_real_l)); */
        /* CUFFT_CHECK(cufftDestroy(plan)); */
      /* } */
  CUDA_CHECK(hipDeviceSynchronize());
}
extern "C" {
void execute_dir_fft_float(float *data_real, hipfftComplex *data_complex,
        int kfield, int *loens, int *offsets, int nfft) {
    execute_fft_new<Float, HIPFFT_R2C>(data_real, data_complex, kfield, loens, offsets, nfft);
}
void execute_inv_fft_float(hipfftComplex *data_complex, float *data_real,
        int kfield, int *loens, int *offsets, int nfft) {
    execute_fft<Float, HIPFFT_C2R>(data_real, data_complex, kfield, loens, offsets, nfft);
}
void execute_dir_fft_double(double *data_real, hipfftDoubleComplex *data_complex,
        int kfield, int *loens, int *offsets, int nfft) {
    execute_fft_new<Double, HIPFFT_D2Z>(data_real, data_complex, kfield, loens, offsets, nfft);
}
void execute_inv_fft_double(hipfftDoubleComplex *data_complex, double *data_real,
        int kfield, int *loens, int *offsets, int nfft) {
    execute_fft<Double, HIPFFT_Z2D>(data_real, data_complex, kfield, loens, offsets, nfft);
}
}
